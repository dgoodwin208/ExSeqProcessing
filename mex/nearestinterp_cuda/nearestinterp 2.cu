#include "hip/hip_runtime.h"
#include <iostream>
#include <cassert>

#include <thrust/copy.h>
#include <thrust/replace.h>
#include <thrust/functional.h>
#include <thrust/iterator/counting_iterator.h>

#include <hip/hip_runtime.h>

#include "nearestinterp.h"
#include "matrix_helper.h"
#include "cuda_timer.h"

#include "spdlog/spdlog.h"


namespace cudautils {

// interpolate image data
//
__global__
void interpolate_volumes(
        unsigned int x_stride,
        unsigned int y_stride,
        unsigned int map_idx_size,
        unsigned int *map_idx,
        int8_t *map,
        double *image,
        double *interpolated_values) {

    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= map_idx_size) return;

    unsigned int idx_zplane = map_idx[i] - 1 - x_stride - (x_stride * y_stride); // move current pos idx by (-1, -1, -1)
    unsigned int idx = idx_zplane;

    int sum_idx = 0;
    double sum = 0.0;

    // (-1, -1, -1)  ->  (1, -1, -1)
    sum += image[idx + 0] * double(map[idx + 0]);   sum_idx += (map[idx + 0] > 0);
    sum += image[idx + 1] * double(map[idx + 1]);   sum_idx += (map[idx + 1] > 0);
    sum += image[idx + 2] * double(map[idx + 2]);   sum_idx += (map[idx + 2] > 0);

    // (-1, 0, -1)  ->  (1, 0, -1)
    idx += x_stride;
    sum += image[idx + 0] * double(map[idx + 0]);   sum_idx += (map[idx + 0] > 0);
    sum += image[idx + 1] * double(map[idx + 1]);   sum_idx += (map[idx + 1] > 0);
    sum += image[idx + 2] * double(map[idx + 2]);   sum_idx += (map[idx + 2] > 0);

    // (-1, 1, -1)  ->  (1, 1, -1)
    idx += x_stride;
    sum += image[idx + 0] * double(map[idx + 0]);   sum_idx += (map[idx + 0] > 0);
    sum += image[idx + 1] * double(map[idx + 1]);   sum_idx += (map[idx + 1] > 0);
    sum += image[idx + 2] * double(map[idx + 2]);   sum_idx += (map[idx + 2] > 0);

    idx_zplane += x_stride * y_stride;
    idx = idx_zplane;

    // (-1, -1, 0)  ->  (1, -1, 0)
    sum += image[idx + 0] * double(map[idx + 0]);   sum_idx += (map[idx + 0] > 0);
    sum += image[idx + 1] * double(map[idx + 1]);   sum_idx += (map[idx + 1] > 0);
    sum += image[idx + 2] * double(map[idx + 2]);   sum_idx += (map[idx + 2] > 0);

    // (-1, 0, 0)  ->  (1, 0, 0)
    idx += x_stride;
    sum += image[idx + 0] * double(map[idx + 0]);   sum_idx += (map[idx + 0] > 0);
    sum += image[idx + 1] * double(map[idx + 1]);   sum_idx += (map[idx + 1] > 0);
    sum += image[idx + 2] * double(map[idx + 2]);   sum_idx += (map[idx + 2] > 0);

    // (-1, 1, 0)  ->  (1, 1, 0)
    idx += x_stride;
    sum += image[idx + 0] * double(map[idx + 0]);   sum_idx += (map[idx + 0] > 0);
    sum += image[idx + 1] * double(map[idx + 1]);   sum_idx += (map[idx + 1] > 0);
    sum += image[idx + 2] * double(map[idx + 2]);   sum_idx += (map[idx + 2] > 0);

    idx_zplane += x_stride * y_stride;
    idx = idx_zplane;

    // (-1, -1, 1)  ->  (1, -1, 1)
    sum += image[idx + 0] * double(map[idx + 0]);   sum_idx += (map[idx + 0] > 0);
    sum += image[idx + 1] * double(map[idx + 1]);   sum_idx += (map[idx + 1] > 0);
    sum += image[idx + 2] * double(map[idx + 2]);   sum_idx += (map[idx + 2] > 0);

    // (-1, 0, 1)  ->  (1, 0, 1)
    idx += x_stride;
    sum += image[idx + 0] * double(map[idx + 0]);   sum_idx += (map[idx + 0] > 0);
    sum += image[idx + 1] * double(map[idx + 1]);   sum_idx += (map[idx + 1] > 0);
    sum += image[idx + 2] * double(map[idx + 2]);   sum_idx += (map[idx + 2] > 0);

    // (-1, 1, 1)  ->  (1, 1, 1)
    idx += x_stride;
    sum += image[idx + 0] * double(map[idx + 0]);   sum_idx += (map[idx + 0] > 0);
    sum += image[idx + 1] * double(map[idx + 1]);   sum_idx += (map[idx + 1] > 0);
    sum += image[idx + 2] * double(map[idx + 2]);   sum_idx += (map[idx + 2] > 0);

    if (sum_idx > 0) {
        interpolated_values[i] = sum / double(sum_idx);
    } else {
        idx_zplane = map_idx[i] - 2 * (1 + x_stride + (x_stride * y_stride)); // move current pos idx by (-2, -2, -2)

        // (u, v, w) <- (x, y, z)
        // u=0-4 v=0-4 w=0,4
        idx = idx_zplane;
        for (unsigned int v = 0; v < 5; v++) {
            sum += image[idx + 0] * double(map[idx + 0]);   sum_idx += (map[idx + 0] > 0);
            sum += image[idx + 1] * double(map[idx + 1]);   sum_idx += (map[idx + 1] > 0);
            sum += image[idx + 2] * double(map[idx + 2]);   sum_idx += (map[idx + 2] > 0);
            sum += image[idx + 3] * double(map[idx + 3]);   sum_idx += (map[idx + 3] > 0);
            sum += image[idx + 4] * double(map[idx + 4]);   sum_idx += (map[idx + 4] > 0);
            idx += x_stride;
        }
        idx = idx_zplane + 4 * x_stride * y_stride;
        for (unsigned int v = 0; v < 5; v++) {
            sum += image[idx + 0] * double(map[idx + 0]);   sum_idx += (map[idx + 0] > 0);
            sum += image[idx + 1] * double(map[idx + 1]);   sum_idx += (map[idx + 1] > 0);
            sum += image[idx + 2] * double(map[idx + 2]);   sum_idx += (map[idx + 2] > 0);
            sum += image[idx + 3] * double(map[idx + 3]);   sum_idx += (map[idx + 3] > 0);
            sum += image[idx + 4] * double(map[idx + 4]);   sum_idx += (map[idx + 4] > 0);
            idx += x_stride;
        }


        // u=0-4 v=0,4 w=1-3
        for (unsigned int w = 1; w < 4; w++) {
            idx = idx_zplane + w * x_stride * y_stride;
            sum += image[idx + 0] * double(map[idx + 0]);   sum_idx += (map[idx + 0] > 0);
            sum += image[idx + 1] * double(map[idx + 1]);   sum_idx += (map[idx + 1] > 0);
            sum += image[idx + 2] * double(map[idx + 2]);   sum_idx += (map[idx + 2] > 0);
            sum += image[idx + 3] * double(map[idx + 3]);   sum_idx += (map[idx + 3] > 0);
            sum += image[idx + 4] * double(map[idx + 4]);   sum_idx += (map[idx + 4] > 0);
            idx += 4 * x_stride;
            sum += image[idx + 0] * double(map[idx + 0]);   sum_idx += (map[idx + 0] > 0);
            sum += image[idx + 1] * double(map[idx + 1]);   sum_idx += (map[idx + 1] > 0);
            sum += image[idx + 2] * double(map[idx + 2]);   sum_idx += (map[idx + 2] > 0);
            sum += image[idx + 3] * double(map[idx + 3]);   sum_idx += (map[idx + 3] > 0);
            sum += image[idx + 4] * double(map[idx + 4]);   sum_idx += (map[idx + 4] > 0);
        }

        // u=0,4 v=1-3 w=1-3
        for (unsigned int w = 1; w < 4; w++) {
            idx = idx_zplane + w * x_stride * y_stride;
            idx += x_stride;
            sum += image[idx + 0] * double(map[idx + 0]);   sum_idx += (map[idx + 0] > 0);
            sum += image[idx + 4] * double(map[idx + 4]);   sum_idx += (map[idx + 4] > 0);
            idx += x_stride;
            sum += image[idx + 0] * double(map[idx + 0]);   sum_idx += (map[idx + 0] > 0);
            sum += image[idx + 4] * double(map[idx + 4]);   sum_idx += (map[idx + 4] > 0);
            idx += x_stride;
            sum += image[idx + 0] * double(map[idx + 0]);   sum_idx += (map[idx + 0] > 0);
            sum += image[idx + 4] * double(map[idx + 4]);   sum_idx += (map[idx + 4] > 0);
        }

        if (sum_idx > 0) {
            interpolated_values[i] = sum / double(sum_idx);
        } else {
            interpolated_values[i] = 0.0;
        }
    }
    return;
}


NearestInterp::NearestInterp(
        const unsigned int x_size,
        const unsigned int y_size,
        const unsigned int z_size,
        const unsigned int x_sub_size,
        const unsigned int y_sub_size,
        const unsigned int dx,
        const unsigned int dy,
        const unsigned int dw,
        const unsigned int num_gpus,
        const unsigned int num_streams)
    : x_size_(x_size), y_size_(y_size), z_size_(z_size),
        x_sub_size_(x_sub_size), y_sub_size_(y_sub_size),
        dx_(dx), dy_(dy), dw_(dw),
        num_gpus_(num_gpus), num_streams_(num_streams),
        subdom_data_(num_gpus) {

    logger_ = spdlog::get("console");
    if (! logger_) {
        logger_ = spdlog::stdout_logger_mt("console");
    }
#ifdef DEBUG_OUTPUT
    spdlog::set_level(spdlog::level::debug);
#else
    spdlog::set_level(spdlog::level::info);
#endif

    size_t log_q_size = 4096;
    spdlog::set_async_mode(log_q_size);

    num_x_sub_ = get_num_blocks(x_size_, x_sub_size_);
    num_y_sub_ = get_num_blocks(y_size_, y_sub_size_);

    x_sub_stride_ = x_sub_size_ + 2 * dw_;
    y_sub_stride_ = y_sub_size_ + 2 * dw_;

    dx_stride_ = dx_ + 2 * dw_;
    dy_stride_ = dy_ + 2 * dw_;
    z_stride_ = z_size_ + 2 * dw_;
#ifdef DEBUG_OUTPUT
    logger_->info("x_size={}, x_sub_size={}, num_x_sub={}, x_sub_stride={}, dx={}, dx_stride={}",
            x_size_, x_sub_size_, num_x_sub_, x_sub_stride_, dx_, dx_stride_);
    logger_->info("y_size={}, y_sub_size={}, num_y_sub={}, y_sub_stride={}, dy={}, dy_stride={}",
            y_size_, y_sub_size_, num_y_sub_, y_sub_stride_, dy_, dy_stride_);
    logger_->info("z_size={}, dw={}, z_stride={}", z_size_, dw_, z_stride_);
#endif


    dom_data_ = std::make_shared<DomainDataOnHost>(x_size_, y_size_, z_size_);

    for (unsigned int i = 0; i < num_gpus_; i++) {
        hipSetDevice(i);

        subdom_data_[i] = std::make_shared<SubDomainDataOnGPU>(x_sub_stride_, y_sub_stride_, z_stride_, num_streams_);

        for (unsigned int j = 0; j < num_streams_; j++) {
            subdom_data_[i]->stream_data[j] = std::make_shared<SubDomainDataOnStream>(dx_stride_, dy_stride_, z_stride_);

            hipStreamCreate(&subdom_data_[i]->stream_data[j]->stream);
        }
    }
    hipSetDevice(0);


    unsigned int idx_gpu = 0;
    for (unsigned int y_sub_i = 0; y_sub_i < num_y_sub_; y_sub_i++) {
        for (unsigned int x_sub_i = 0; x_sub_i < num_x_sub_; x_sub_i++) {
            subdom_data_[idx_gpu]->x_sub_i_list.push_back(x_sub_i);
            subdom_data_[idx_gpu]->y_sub_i_list.push_back(y_sub_i);

            idx_gpu++;
            if (idx_gpu == num_gpus) {
                idx_gpu = 0;
            }
        }
    }

}

NearestInterp::~NearestInterp() {
    for (unsigned int i = 0; i < num_gpus_; i++) {
        for (unsigned int j = 0; j < num_streams_; j++) {
            hipStreamDestroy(subdom_data_[i]->stream_data[j]->stream);
        }
    }

    //logger_->flush();
}

void NearestInterp::setImage(const double *img)
{
    thrust::copy(img, img + (x_size_ * y_size_ * z_size_), dom_data_->h_image);
}

void NearestInterp::setImage(const std::vector<double>& img)
{
    assert((x_size_ * y_size_ * z_size_) == img.size());

    thrust::copy(img.begin(), img.end(), dom_data_->h_image);
}

void NearestInterp::setMapToBeInterpolated(const int8_t *map)
{
    thrust::copy(map, map + (x_size_ * y_size_ * z_size_), dom_data_->h_map);
}

void NearestInterp::setMapToBeInterpolated(const std::vector<int8_t>& map)
{
    assert((x_size_ * y_size_ * z_size_) == map.size());

    thrust::copy(map.begin(), map.end(), dom_data_->h_map);
}

void NearestInterp::getImage(double *img)
{
    thrust::copy(dom_data_->h_image, dom_data_->h_image + x_size_ * y_size_ * z_size_, img);
}

void NearestInterp::getImage(std::vector<double>& img)
{
    thrust::copy(dom_data_->h_image, dom_data_->h_image + x_size_ * y_size_ * z_size_, img.begin());
}


int NearestInterp::getNumOfGPUTasks(const int gpu_id) {
    return subdom_data_[gpu_id]->x_sub_i_list.size();
}

int NearestInterp::getNumOfStreamTasks(
        const int gpu_id,
        const int stream_id) {
    return 1;
}

void NearestInterp::postrun() {
    for (int gpu_id = 0; gpu_id < num_gpus_; gpu_id++) {
        std::shared_ptr<SubDomainDataOnGPU> subdom_data = subdom_data_[gpu_id];

        for (int stream_id = 0; stream_id < num_streams_; stream_id++) {
            std::shared_ptr<SubDomainDataOnStream> stream_data = subdom_data->stream_data[stream_id];

            assert(stream_data->interpolated_map_idx.size() == stream_data->interpolated_values.size());
            for (size_t i = 0; i < stream_data->interpolated_map_idx.size(); i++) {
                size_t idx = stream_data->interpolated_map_idx[i];
                double val = stream_data->interpolated_values[i];
                dom_data_->h_image[idx] = val;
            }
        }
    }
}

void NearestInterp::runOnGPU(
        const int gpu_id,
        const unsigned int gpu_task_id) {

    hipSetDevice(gpu_id);

    std::shared_ptr<SubDomainDataOnGPU> subdom_data = subdom_data_[gpu_id];
    std::shared_ptr<SubDomainDataOnStream> stream_data0 = subdom_data->stream_data[0];

    unsigned int x_sub_i = subdom_data->x_sub_i_list[gpu_task_id];
    unsigned int y_sub_i = subdom_data->y_sub_i_list[gpu_task_id];
#ifdef DEBUG_OUTPUT
    CudaTimer timer;
    logger_->info("===== gpu_id={} x_sub_i={} y_sub_i={}", gpu_id, x_sub_i, y_sub_i);
#endif

    unsigned int x_sub_start = x_sub_i * x_sub_size_;
    unsigned int x_sub_delta = get_delta(x_size_, x_sub_i, x_sub_size_);
    unsigned int y_sub_start = y_sub_i * y_sub_size_;
    unsigned int y_sub_delta = get_delta(y_size_, y_sub_i, y_sub_size_);
    unsigned int base_x_sub  = (x_sub_i > 0 ? 0 : dw_);
    unsigned int base_y_sub  = (y_sub_i > 0 ? 0 : dw_);

    unsigned int padding_x_sub_start = x_sub_start - (x_sub_i > 0 ? dw_ : 0);
    unsigned int padding_x_sub_delta = x_sub_delta + (x_sub_i > 0 ? dw_ : 0) + (x_sub_i < num_x_sub_ - 1 ? dw_ : 0);
    unsigned int padding_y_sub_start = y_sub_start - (y_sub_i > 0 ? dw_ : 0);
    unsigned int padding_y_sub_delta = y_sub_delta + (y_sub_i > 0 ? dw_ : 0) + (y_sub_i < num_y_sub_ - 1 ? dw_ : 0);
#ifdef DEBUG_OUTPUT
    unsigned int x_sub_end = x_sub_start + x_sub_delta;
    unsigned int y_sub_end = y_sub_start + y_sub_delta;
    logger_->debug("x_sub=({},{},{}) y_sub=({},{},{})", x_sub_start, x_sub_delta, x_sub_end, y_sub_start, y_sub_delta, y_sub_end);
    logger_->debug("base_x_sub={},base_y_sub={}", base_x_sub, base_y_sub);
#endif

    size_t padded_sub_volume_size = x_sub_stride_ * y_sub_stride_ * z_stride_;

    int8_t *padded_sub_map;
    double *padded_sub_image;
    hipHostAlloc(&padded_sub_map,   padded_sub_volume_size * sizeof(int8_t), hipHostMallocPortable);
    hipHostAlloc(&padded_sub_image, padded_sub_volume_size * sizeof(double), hipHostMallocPortable);

    thrust::fill(padded_sub_map, padded_sub_map + padded_sub_volume_size, -1);

    for (unsigned int k = 0; k < z_size_; k++) {
        for (unsigned int j = 0; j < padding_y_sub_delta; j++) {
            size_t src_idx = dom_data_->sub2ind(padding_x_sub_start, padding_y_sub_start + j, k);
            size_t dst_idx = subdom_data->pad_sub2ind(base_x_sub, base_y_sub + j, dw_ + k);

            int8_t* src_map_begin = &(dom_data_->h_map[src_idx]);
            int8_t* dst_map_begin = &(padded_sub_map[dst_idx]);
            thrust::copy(src_map_begin, src_map_begin + padding_x_sub_delta, dst_map_begin);

            double* src_image_begin = &(dom_data_->h_image[src_idx]);
            double* dst_image_begin = &(padded_sub_image[dst_idx]);
            thrust::copy(src_image_begin, src_image_begin + padding_x_sub_delta, dst_image_begin);
        }
    }

    thrust::fill(thrust::device, subdom_data->padded_image, subdom_data->padded_image + padded_sub_volume_size, 0.0);

    hipMemcpyAsync(
            subdom_data->padded_image,
            padded_sub_image,
            padded_sub_volume_size * sizeof(double),
            hipMemcpyHostToDevice, stream_data0->stream);

#ifdef DEBUG_OUTPUT
    hipStreamSynchronize(stream_data0->stream);
    logger_->info("transfer image data {}", timer.get_laptime());

#ifdef DEBUG_OUTPUT_MATRIX
    logger_->info("===== dev image");
    print_matrix3d(logger_, x_size_, y_size_, 0, 0, 0, x_size_, y_size_, z_size_, dom_data_->h_image);
    print_matrix3d_dev(logger_, x_sub_stride_, y_sub_stride_, z_stride_, 0, 0, 0, x_sub_stride_, y_sub_stride_, z_stride_, subdom_data->padded_image);
#endif

    timer.reset();
#endif

    hipMemcpyAsync(
            subdom_data->padded_map,
            padded_sub_map,
            padded_sub_volume_size * sizeof(int8_t),
            hipMemcpyHostToDevice, stream_data0->stream);

#ifdef DEBUG_OUTPUT
    hipStreamSynchronize(stream_data0->stream);
    logger_->info("transfer map data {}", timer.get_laptime());

#ifdef DEBUG_OUTPUT_MATRIX
    logger_->debug("===== dev map");
    print_matrix3d(logger_, x_size_, y_size_, 0, 0, 0, x_size_, y_size_, z_size_, dom_data_->h_map);
    print_matrix3d_dev(logger_, x_sub_stride_, y_sub_stride_, z_stride_, 0, 0, 0, x_sub_stride_, y_sub_stride_, z_stride_, subdom_data->padded_map);
#endif

    timer.reset();
#endif

    thrust::fill(thrust::device, subdom_data->padded_map_idx, subdom_data->padded_map_idx + padded_sub_volume_size, 0.0);

    auto end_itr = thrust::copy_if(
            thrust::device,
            thrust::make_counting_iterator<unsigned int>(0),
            thrust::make_counting_iterator<unsigned int>(padded_sub_volume_size),
            subdom_data->padded_map,
            subdom_data->padded_map_idx,
            thrust::logical_not<int8_t>());

    subdom_data->padded_map_idx_size = end_itr - subdom_data->padded_map_idx;

    thrust::replace(thrust::device, subdom_data->padded_map, subdom_data->padded_map + padded_sub_volume_size, -1, 0);

#ifdef DEBUG_OUTPUT
    hipStreamSynchronize(stream_data0->stream);
    logger_->info("calculate map idx {}", timer.get_laptime());

    logger_->info("padded_map_idx_size={}", subdom_data->padded_map_idx_size);
    logger_->debug("===== padded_map idx");
    thrust::host_vector<unsigned int> dbg_padded_map_idx(thrust::device_vector<unsigned int>(subdom_data->padded_map_idx, subdom_data->padded_map_idx + subdom_data->padded_map_idx_size));
    std::copy(dbg_padded_map_idx.begin(), dbg_padded_map_idx.end(), std::ostream_iterator<unsigned int>(std::cout, ","));
    std::cout << std::endl;

    timer.reset();
#endif

    for (int i = 0; i < num_streams_; i++) {
        subdom_data->stream_data[i]->dx_i_list.clear();
        subdom_data->stream_data[i]->dy_i_list.clear();
    }

    unsigned int num_dx = get_num_blocks(x_sub_delta, dx_);
    unsigned int num_dy = get_num_blocks(y_sub_delta, dy_);
    unsigned int stream_id = 0;
    for (unsigned int dy_i = 0; dy_i < num_dy; dy_i++) {
        for (unsigned int dx_i = 0; dx_i < num_dx; dx_i++) {
            subdom_data->stream_data[stream_id]->dx_i_list.push_back(dx_i);
            subdom_data->stream_data[stream_id]->dy_i_list.push_back(dy_i);

            stream_id++;
            if (stream_id == num_streams_) {
                stream_id = 0;
            }
        }
    }
    hipStreamSynchronize(stream_data0->stream);

    hipHostFree(padded_sub_map);
    hipHostFree(padded_sub_image);
}

void NearestInterp::runOnStream(
        const int gpu_id,
        const int stream_id,
        const unsigned int gpu_task_id) {

    hipSetDevice(gpu_id);

    std::shared_ptr<SubDomainDataOnGPU> subdom_data = subdom_data_[gpu_id];
    std::shared_ptr<SubDomainDataOnStream> stream_data = subdom_data->stream_data[stream_id];

    unsigned int x_sub_i = subdom_data->x_sub_i_list[gpu_task_id];
    unsigned int y_sub_i = subdom_data->y_sub_i_list[gpu_task_id];
    unsigned int x_sub_delta = get_delta(x_size_, x_sub_i, x_sub_size_);
    unsigned int y_sub_delta = get_delta(y_size_, y_sub_i, y_sub_size_);
    unsigned int x_sub_start = x_sub_i * x_sub_size_;
    unsigned int y_sub_start = y_sub_i * y_sub_size_;

#ifdef DEBUG_OUTPUT
    CudaTimer timer(stream_data->stream);
#endif

    for (auto dx_itr = stream_data->dx_i_list.begin(), dy_itr = stream_data->dy_i_list.begin();
            dx_itr != stream_data->dx_i_list.end() || dy_itr != stream_data->dy_i_list.end(); dx_itr++, dy_itr++) {

        unsigned int dx_i = *dx_itr;
        unsigned int dy_i = *dy_itr;

        unsigned int dx_start = dx_i * dx_;
        unsigned int dx_delta = get_delta(x_sub_delta, dx_i, dx_);
        unsigned int dx_end   = dx_start + dx_delta;
        unsigned int dy_start = dy_i * dy_;
        unsigned int dy_delta = get_delta(y_sub_delta, dy_i, dy_);
        unsigned int dy_end   = dy_start + dy_delta;

#ifdef DEBUG_OUTPUT
        logger_->info("dx_i={}, dy_i={}", dx_i, dy_i);
        logger_->info("x=({},{},{}) y=({},{},{}), dw={}", dx_start, dx_delta, dx_end, dy_start, dy_delta, dy_end, dw_);
        logger_->info("padded_map_idx_size={}", subdom_data->padded_map_idx_size);

        logger_->debug("===== all padded_map idx");
        thrust::host_vector<unsigned int> dbg_all_padded_map_idx(thrust::device_vector<unsigned int>(subdom_data->padded_map_idx, subdom_data->padded_map_idx + subdom_data->padded_map_idx_size));
        std::copy(dbg_all_padded_map_idx.begin(), dbg_all_padded_map_idx.end(), std::ostream_iterator<unsigned int>(std::cout, ","));
        std::cout << std::endl;
#endif


        unsigned int *padded_map_idx;
        hipMalloc(&padded_map_idx, subdom_data->padded_map_idx_size * sizeof(unsigned int));

        RangeCheck range_check { x_sub_stride_, y_sub_stride_,
            dx_start + dw_, dx_end + dw_, dy_start + dw_, dy_end + dw_, dw_, z_size_ + dw_ };

        auto end_itr = thrust::copy_if(
                thrust::device,
                subdom_data->padded_map_idx,
                subdom_data->padded_map_idx + subdom_data->padded_map_idx_size,
                padded_map_idx,
                range_check);

        unsigned int padded_map_idx_size = end_itr - padded_map_idx;

#ifdef DEBUG_OUTPUT
        logger_->info("padded_map_idx_size={}", padded_map_idx_size);
        logger_->info("transfer map idx {}", timer.get_laptime());

        hipStreamSynchronize(stream_data->stream);

        thrust::host_vector<unsigned int> dbg_h_padded_map_idx(thrust::device_vector<unsigned int>(padded_map_idx, padded_map_idx + padded_map_idx_size));
        for (unsigned int i = 0; i < padded_map_idx_size; i++) {
            logger_->debug("padded_map_idx={}", dbg_h_padded_map_idx[i]);
        }
        timer.reset();
#endif
        if (padded_map_idx_size == 0) {
#ifdef DEBUG_OUTPUT
            logger_->debug("no map to be padded");
#endif
            continue;
        }

        double *interpolated_values;
        hipMalloc(&interpolated_values, padded_map_idx_size * sizeof(double));


        unsigned int num_blocks = get_num_blocks(padded_map_idx_size, 1024);
#ifdef DEBUG_OUTPUT
        logger_->info("num_blocks={}", num_blocks);
#endif

        interpolate_volumes<<<num_blocks, 1024, 0, stream_data->stream>>>(
                x_sub_stride_, y_sub_stride_, padded_map_idx_size,
                padded_map_idx,
                subdom_data->padded_map,
                subdom_data->padded_image,
                interpolated_values);

#ifdef DEBUG_OUTPUT
        logger_->info("interpolate volumes {}", timer.get_laptime());

        //debug
//        hipStreamSynchronize(stream_data->stream);
//        std::copy(interpolated_values.begin(),
//                  interpolated_values.begin() + padded_map_idx_size,
//                  std::ostream_iterator<double>(std::cout, ","));
//        std::cout << std::endl;

        timer.reset();
#endif

        double *h_interpolated_values;
        hipHostAlloc(&h_interpolated_values, padded_map_idx_size * sizeof(double), hipHostMallocPortable);

        hipMemcpyAsync(
                h_interpolated_values,
                interpolated_values,
                padded_map_idx_size * sizeof(double),
                hipMemcpyDeviceToHost, stream_data->stream);

        unsigned int *h_padded_map_idx;
        hipHostAlloc(&h_padded_map_idx, padded_map_idx_size * sizeof(unsigned int), hipHostMallocPortable);

        hipMemcpyAsync(
                h_padded_map_idx,
                padded_map_idx,
                padded_map_idx_size * sizeof(unsigned int),
                hipMemcpyDeviceToHost, stream_data->stream);

        hipStreamSynchronize(stream_data->stream);
        for (unsigned int i = 0; i < padded_map_idx_size; i++) {
            unsigned int padding_x;
            unsigned int padding_y;
            unsigned int padding_z;
            ind2sub(x_sub_stride_, y_sub_stride_, h_padded_map_idx[i], padding_x, padding_y, padding_z);
            size_t idx = dom_data_->sub2ind(x_sub_start + padding_x - dw_, y_sub_start + padding_y - dw_, padding_z - dw_);

            stream_data->interpolated_map_idx.push_back(idx);
            stream_data->interpolated_values.push_back(h_interpolated_values[i]);
        }

        hipFree(padded_map_idx);
        hipFree(interpolated_values);

        hipHostFree(h_interpolated_values);
        hipHostFree(h_padded_map_idx);

#ifdef DEBUG_OUTPUT
        logger_->info("transfer d2h and copy interpolated values {}", timer.get_laptime());

#ifdef DEBUG_OUTPUT_MATRIX
        logger_->debug("===== host interp image");
        print_matrix3d(logger_, x_size_, y_size_, 0, 0, 0, x_size_, y_size_, z_size_, dom_data_->h_image);
#endif
#endif
    }
}


} // namespace cudautils

