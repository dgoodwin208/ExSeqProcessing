#include <cstdint>
#include <cassert>

#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <thrust/system/tbb/execution_policy.h>

#include "radixsort.h"

namespace cudautils {

template <typename T1, typename T2>
void radixsort(std::vector<T1>& array1, std::vector<T2>& array2)
{
    assert(array1.size() == array2.size());
    size_t array_size = array1.size();

    hipError_t err;
    T1 *d_keys;
    T2 *d_values;
    err = hipMalloc(&d_keys,   array_size * sizeof(T1));
    if (err == hipErrorOutOfMemory) {
        throw std::bad_alloc();
    }
    err= hipMalloc(&d_values, array_size * sizeof(T2));
    if (err == hipErrorOutOfMemory) {
        throw std::bad_alloc();
    }

    hipMemcpy(d_keys,   array1.data(), array_size * sizeof(T1), hipMemcpyHostToDevice);
    hipMemcpy(d_values, array2.data(), array_size * sizeof(T2), hipMemcpyHostToDevice);

    thrust::sort_by_key(thrust::device, d_keys, d_keys + array_size, d_values);

    hipMemcpy(array1.data(), d_keys,   array_size * sizeof(T1), hipMemcpyDeviceToHost);
    hipMemcpy(array2.data(), d_values, array_size * sizeof(T2), hipMemcpyDeviceToHost);

    hipFree(d_keys);
    hipFree(d_values);
}

template
void radixsort<uint16_t, unsigned int>(std::vector<uint16_t>& array1, std::vector<unsigned int>& array2);

template
void radixsort<unsigned int, double>(std::vector<unsigned int>& array1, std::vector<double>& array2);

template
void radixsort<unsigned int, float>(std::vector<unsigned int>& array1, std::vector<float>& array2);

}

